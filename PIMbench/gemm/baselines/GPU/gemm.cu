/* File:     gemm.cu
 * Purpose:  Implement gemm on a gpu
 *
 */
// TODO: support different data type

#include <hipblas.h>
#include <iostream>
#include <nvml.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "utilBaselines.h"

using namespace std;

#define TOLERANCE 200.0f

vector<float> A;
vector<float> B;
vector<float> C;

// Params ---------------------------------------------------------------------
typedef struct Params {
  uint64_t row, column_A, column_B;
} Params;

void usage() {
  fprintf(stderr, "\nUsage:  ./gemm.out [options]"
                  "\n"
                  "\n    -r <R>    row size"
                  "\n    -c <C>    MatA column size"
                  "\n    -d <C>    MatB column size"
                  "\n");
}

struct Params input_params(int argc, char **argv) {
  struct Params p;
  p.row = 1024;
  p.column_A = 1024;
  p.column_B = 1024;

  int opt;
  while ((opt = getopt(argc, argv, ":r:c:d:")) >= 0) {
    switch (opt) {
    case 'h':
      usage();
      exit(0);
      break;
    case 'r':
      p.row = atoll(optarg);
      break;
    case 'c':
      p.column_A = atoll(optarg);
      break;
    case 'd':
      p.column_B = atoll(optarg);
      break;
    default:
      fprintf(stderr, "\nUnrecognized option!\n");
      usage();
      exit(0);
    }
  }
  return p;
}

int main(int argc, char *argv[]) {
  struct Params p = input_params(argc, argv);

  uint64_t row = p.row, col_A = p.column_A, col_B = p.column_B;

  getVector(row * col_A, A);
  getVector(col_A * col_B, B);

  float *x, *y, *z;

  hipError_t errorCode;

  errorCode = hipMalloc(&x, row * col_A * sizeof(int32_t));
  if (errorCode != hipSuccess) {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }
  errorCode = hipMalloc(&y, col_A * col_B * sizeof(int32_t));
  if (errorCode != hipSuccess) {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }
  errorCode = hipMalloc(&z, row * col_B * sizeof(int32_t));
  if (errorCode != hipSuccess) {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemcpy(x, A.data(), row * col_A * sizeof(float),
                         hipMemcpyHostToDevice);
  if (errorCode != hipSuccess) {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  errorCode = hipMemcpy(y, B.data(), col_A * col_B * sizeof(float),
                         hipMemcpyHostToDevice);
  if (errorCode != hipSuccess) {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }

  const float alpha = 1.0;
  const float beta = 1.0;
  hipblasHandle_t handle;
  hipblasStatus_t status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS initialization failed\n";
    exit(1);
  }

  // **Get active CUDA device**
  int cudaDevice;
  errorCode = hipGetDevice(&cudaDevice);
  if (errorCode != hipSuccess) {
    cerr << "Cuda Error: " << hipGetErrorString(errorCode) << "\n";
    exit(1);
  }
  nvmlReturn_t result;
  nvmlDevice_t device;
  result = nvmlInit();
  if (result != NVML_SUCCESS) {
    std::cerr << "Failed to initialize NVML: " << nvmlErrorString(result)
              << std::endl;
    return 1;
  }

  result = nvmlDeviceGetHandleByIndex(cudaDevice, &device);
  if (result != NVML_SUCCESS) {
    std::cerr << "Failed to get GPU handle: " << nvmlErrorString(result)
              << std::endl;
    return 1;
  }

  // Variables for power sampling
  std::vector<unsigned int> powerSamples;

  // Event creation
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float timeElapsed = 0;

  // Start timer
  hipEventRecord(start, 0);
  /* Kernel Call */
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row, col_B, col_A, &alpha, x,
              row, y, col_A, &beta, z, row);

  while (true) {
    unsigned int power;
    if (nvmlDeviceGetPowerUsage(device, &power) == NVML_SUCCESS) {
      powerSamples.push_back(power);
    }
    if (hipEventQuery(stop) == hipSuccess) {
      break;
    }
  }

  // End timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timeElapsed, start, stop);

  double totalPower = 0;
  for (size_t i = 0; i < powerSamples.size(); ++i) {
    totalPower += powerSamples[i]; // Convert mW to W * time
  }

  float avgPower_mW = totalPower / powerSamples.size(); // Average power in mW

  // **Compute Energy in milliJoules (mJ)**
  float energy_mJ = avgPower_mW * timeElapsed / 1000;

  printf("Execution time for gemm = %f ms\n", timeElapsed);
  printf("Average Power = %f mW\n", avgPower_mW);
  printf("Energy Consumption = %f mJ\n", energy_mJ);

  /* Free memory */
  hipFree(x);
  hipFree(y);
  hipFree(z);

  return 0;
} /* main */
